#include "hip/hip_runtime.h"
﻿//FILTRU PREWITT PE AXA Y - MILEA DANIEL - CRISTIAN

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#define WIDTH 256
#define HEIGHT 256
#define FILTER_SIZE 3

__global__ void prewittFilterY(int* input, int* output, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int col = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if (row < height - 1 && col < width - 1) {
        int sum = 0;
        for (int i = -1; i <= 1; ++i) {
            for (int j = -1; j <= 1; ++j) {
                int rowIdx = row + i;
                int colIdx = col + j;

                int filterVal = (i == -1) ? -1 : ((i == 1) ? 1 : 0);

                sum += input[rowIdx * width + colIdx] * filterVal;
            }
        }
        output[row * width + col] = sum;
    }
}

int main() {
    int* hostInput, * hostOutput, * deviceInput, * deviceOutput;
    int size = WIDTH * HEIGHT * sizeof(int);

    // Alocare memorie host
    hostInput = (int*)malloc(size);
    hostOutput = (int*)malloc(size);

    // Initializare matrice
    for (int i = 0; i < HEIGHT; ++i) {
        for (int j = 0; j < WIDTH; ++j) {
            hostInput[i * WIDTH + j] = j % WIDTH;
        }
    }

    // Alocare memorie device
    hipMalloc((void**)&deviceInput, size);
    hipMalloc((void**)&deviceOutput, size);

    // copy date de la host la device
    hipMemcpy(deviceInput, hostInput, size, hipMemcpyHostToDevice);

    // 32x32 block si grid dimensiuni
    dim3 blockDim(32, 32);
    dim3 gridDim((WIDTH - 2 + blockDim.x - 1) / blockDim.x, (HEIGHT - 2 + blockDim.y - 1) / blockDim.y);

    // lansare kernel si afisare timp executie
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    prewittFilterY << <gridDim, blockDim >> > (deviceInput, deviceOutput, WIDTH, HEIGHT);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // copy rezultate de la device la host
    hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);

    // afisare
    for (int i = 0; i < HEIGHT; ++i) {
        for (int j = 0; j < WIDTH; ++j) {
            printf("%d ", hostOutput[i * WIDTH + j]);
        }
        printf("\n");
    }

    printf("\nExecution Time: %f ms\n", milliseconds);

    //eliberare memorie
    free(hostInput);
    free(hostOutput);
    hipFree(deviceInput);
    hipFree(deviceOutput);

    return 0;
}
